
#define HIPCUB_STDERR
#include <iostream>
#include <cub/util_allocator.cuh>
#include <cub/device/device_reduce.cuh>
#include <cub/util_debug.cuh>
#include "util.h"

#define T float

using namespace hipcub;

using KVT = KeyValuePair<int, T>;

void init(T *h_in, int num_item){
  for(int i = 0;i < num_item;i ++) {
    h_in[i] = i;
  }
}

KVT* get_out(KVT* d_data, size_t num) {
  KVT* ret = (KVT*)malloc(sizeof(KVT) * num);
  CubDebugExit(hipMemcpy(ret, d_data, sizeof(KVT) * num, hipMemcpyDeviceToHost));
  return ret;
}

int main(int argc, char **argv) {

  int bsize = 1;
  int vsize = 40000;
  int num_item = bsize * vsize;
  int warm = 10;
  int repeat = 100000;

  T * h_in = new T[num_item];
  init(h_in, num_item);

  // Allocate problem device arrays
  T *d_in = NULL;

  CubDebugExit(hipMalloc((void**)&d_in, sizeof(T) * num_item));

  // Init device input
  CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * num_item, hipMemcpyHostToDevice));

  // Alloc device output array
  KVT *d_out = NULL;
  CubDebugExit(hipMalloc((void**)&d_out, sizeof(KVT) * num_item));

  // Request and allocate temporary storage
  void *d_tmp = NULL;
  size_t tmp_bytes = 0;
  CubDebugExit(DeviceReduce::ArgMax(d_tmp, tmp_bytes, d_in, d_out, num_item));
  CubDebugExit(hipMalloc((void**)&d_tmp, tmp_bytes));

  // RUN
  for(int i = 0;i < warm;i ++) {
    CubDebugExit(DeviceReduce::ArgMax(d_tmp, tmp_bytes, d_in, d_out, num_item));
    CubDebugExit(hipDeviceSynchronize());
  }

  std::cout << " WARM OK " << std::endl;

  uint64_t t0, t1;
  t0 = get_time();
  for(int i = 0;i < repeat;i ++) {
    CubDebugExit(DeviceReduce::ArgMax(d_tmp, tmp_bytes, d_in, d_out, num_item));
    CubDebugExit(hipDeviceSynchronize());
  }
  t1 = get_time();
  std::cout << (t1-t0) << " " << (t1 - t0)/repeat << " us." << std::endl;



  KVT* d_ret = get_out(d_out, 1);

  std::cout << d_ret->key << " " << d_ret->value << std::endl;

  CubDebugExit(hipDeviceSynchronize());

  return 0;
}
